#include <stdio.h>

#include <hip/hip_runtime.h>


#include <sys/time.h>

void checkErrors(hipError_t err, const char *msg)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "%s", msg);
    fprintf(stderr,
            " [Erro CUDA: %s]\n",
            hipGetErrorString(err));
    exit(-1);
  }
}

void compareResults(float *C1, float *C2, int numElements)
{
  float epislon = 0.00001;
  for (int i = 0; i < numElements; i++)
  {
    // printf("%d %f %f %f %f %d\n", i, C1[i], C2[i], abs(C1[i] - C2[i]), epislon, abs(C1[i] - C2[i]) > epislon);
    if (abs(C1[i] - C2[i]) > epislon)
    {
      printf("Comparação de resultados falhou\n");
      // printf("%d %f %f %f %f %d\n",i, C1[i], C2[i], abs(C1[i] - C2[i]), epislon, abs(C1[i] - C2[i]) > epislon);
      exit(-1);
    }
  }
}

// void compareResultsN()

__global__ void vecAdd(float *A, float *B, float *C, int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  // printf("i = %d\n", i);
  if (i < numElements)
  {
    // printf("-i = %d\n", i);

    C[i] = A[i] + B[i];
  }
}

__global__ void vecAddN(float *A, float *B, float *C, int numElements)
{
  // blockDim.x é o index do grid. no caso é sempre zero.
  // blockIdx é o index do block. no caso é 0 até numBlocks-1
  // threadIdx.x é o index da thread dentro do bloco. No caso é 0 até maxBlockSize -1
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  printf("i = %d", i);
  if (i < numElements)
  {
    C[i] = A[i] + B[i];
  }
}

void vecAddCPU(float *A, float *B, float *C, int numElements)
{
  for (int i = 0; i < numElements; i++)
  {
    C[i] = A[i] + B[i];
  }
}

int main(int argc, char **argv)
{
  int maxBlockSize = 16*32; // padrão da arquitetura
  int numElements = 324000000;
  // int numberOfSequencialOperations = 4;

  int size = numElements * sizeof(float);
  struct timeval GPU_start, GPU_end, CPU_start, CPU_end;

  printf("Inicializando vetores no host\n");

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);

  if (h_A == NULL || h_B == NULL || h_C == NULL)
  {
    fprintf(stderr, "Erro no malloc do host\n");
    exit(-1);
  }

  // Inicializa vetores de entrada no host
  for (int i = 0; i < numElements; i++)
  {
    h_A[i] = rand() / (float)(RAND_MAX);
    h_B[i] = rand() / (float)(RAND_MAX);
  }

  // Variáveis do device
  float *d_A;
  float *d_B;
  float *d_C;

  printf("Inicializando vetores no device\n");

  gettimeofday(&GPU_start, NULL);

  hipMalloc(&d_A, size); // atenção que cudaMalloc tem interce distinta do malloc da CPU. Vai e não volta.
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  checkErrors(hipGetLastError(), "Malloc nos vetores do device");

  printf("Copiando memória do host para o device\n");
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  checkErrors(hipGetLastError(), "Cópia para o device");

  // printf("   Número de operações sequenciais %d\n", numberOfSequencialOperations);
  // Calculando o número de blocks
  int numBlocks = (numElements + maxBlockSize - 1) / maxBlockSize; //Isso é tipo uma maneira truncar o resultado com segurança?
  // int numBlocks = ((numElements / numberOfSequencialOperations) + maxBlockSize - 1) / maxBlockSize; // Isso é tipo uma maneira truncar o resultado com segurança?

  printf("Lançando um kernel com %d threads, com %d blocks de tamanho %d\n",
         numBlocks * maxBlockSize,
         numBlocks,
         maxBlockSize);

  vecAdd<<<numBlocks, maxBlockSize>>>(d_A, d_B, d_C, numElements);
  // vecAddN<<<numBlocks, maxBlockSize>>>(d_A, d_B, d_C, numElements);

  checkErrors(hipGetLastError(), "Lançamento do kernel");

  printf("Copiando memória do device para host\n");
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  checkErrors(hipGetLastError(), "Cópia para host");

  gettimeofday(&GPU_end, NULL);
  double GPU_elapsed_time = (GPU_end.tv_sec - GPU_start.tv_sec) +
                            (GPU_end.tv_usec - GPU_start.tv_usec) / 1000000.0;
  printf("GPU: %.4fs\n", GPU_elapsed_time);

  printf("Alocando vetor de teste no host\n");
  gettimeofday(&CPU_start, NULL);
  float *h_D = (float *)malloc(size);
  if (h_D == NULL)
  {
    fprintf(stderr, "Erro no malloc do host\n");
    exit(-1);
  }

  printf("Lançando cálculo na CPU\n");
  vecAddCPU(h_A, h_B, h_D, numElements);

  gettimeofday(&CPU_end, NULL);
  double CPU_elapsed_time = (CPU_end.tv_sec - CPU_start.tv_sec) +
                            (CPU_end.tv_usec - CPU_start.tv_usec) / 1000000.0;
  printf("CPU: %.4fs\n", CPU_elapsed_time);

  printf("Comparando resultados entre CPU e GPU\n");
  compareResults(h_C, h_D, numElements);

  printf("Liberando memória\n");
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_D);

  printf("Fim\n");

  return 0;
}